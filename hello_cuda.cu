#include ""
#include <iostream>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for(int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t devProp;

        hipGetDeviceProperties(&devProp, i);
        std::cout << "using GPU device" << i << ":" << devProp.name << std::endl;
        std::cout << "设备全局内存总量:   " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        std::cout << "SM的数量:" << devProp.multiProcessorCount << std::endl;
        std::cout << "每个线程块的共享内存大小: " << devProp.sharedMemPerBlock / 1024.0 << "KB" << std::endl;
        std::cout << "每个线程块的最大线程数：" << devProp.maxThreadPerBlock << std::endl;
        std::cout << "设备上一个线程块（Block）中可用的32位寄存器的数量: " << devProp.regsPerBlock << std::endl;
        std::cout << "每个EM的最大线程数： " << devProp.maxThreadPerMultiProcessor << std::endl;
        std::cout << "每个EM的最大线程束数： " << devProp.maxThreadPerMultiProcessor / 32 << std::endl;
        std::cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << std::endl;

    }
    return 0;
}